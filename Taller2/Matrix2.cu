#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


__host__
void read(float *M, FILE *source, int rows, int cols){
	for (int i = 0; i < rows; ++i){
		for (int j = 0; j < cols; ++j){
			fscanf(source, "%f,", &M[i * cols + j]);
		}
	}
	fclose(source);
	return;
}

__host__
void print(float *M, int rows, int cols){
  printf("\n");
  printf("----------------------------------------\n");
  for(int i = 0; i < rows; i++) {
  		for(int j = 0; j < cols; j++) {
     		printf("%.2f ", M[i * cols + j]);
    	}
		printf("\n");
  }
  printf("----------------------------------------\n");
  printf("\n");
  return;
}

__global__
void MatrixMultiplyKernel(float *d_A, float *d_B, float *d_R, int colsA, int rowsA, int colsB, int rowsB){

	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if((row < rowsA) && (col < colsB)){
		float cont = 0.0;
		for (int k = 0; k < rowsB; ++k){
			cont += d_A[row * colsA + k] * d_B[k * colsB + col];
		}
		d_R[row * colsB + col] = cont;
	}
	return;
}


int main(int argc, char** argv)
{

	if (argc != 3){
		printf("Debe añadir los nombres de los archivos\n");
		return 1;
	}

	float *h_A, *h_B, *h_R;
	int rowsA, rowsB, colsA, colsB;


	hipError_t error = hipSuccess;

	FILE *file_1, *file_2;
	file_1 = fopen(argv[1], "r");
	file_2 = fopen(argv[2], "r");

	fscanf(file_1, "%d", &rowsA);
	fscanf(file_1, "%d", &colsA);
	fscanf(file_2, "%d", &rowsB);
	fscanf(file_2, "%d", &colsB);

	if (colsA != rowsB){
		printf("Es imposible multiplicar las matrices\n");
		return 1;
	}

	float sizeA = rowsA * colsA * sizeof(float);
	float sizeB = rowsB * colsB * sizeof(float);
	float sizeR = rowsA * colsB * sizeof(float);


	h_A = (float*)malloc(sizeA);
	h_B = (float*)malloc(sizeB);
	h_R = (float*)malloc(sizeR);

	read(h_A, file_1, rowsA, colsA);
	read(h_B, file_2, rowsB, colsB);

	float *d_A, *d_B, *d_R;

	error = hipMalloc((void**)&d_A, sizeA);
	if (error != hipSuccess){
		printf("Error solicitando memoria para d_A \n");
		return 1;
	}

	error = hipMalloc((void**)&d_B, sizeB);
	if (error != hipSuccess){
		printf("Error solicitando memoria para d_B \n");
		return 1;
	}

	error = hipMalloc((void**)&d_R, sizeR);
	if (error != hipSuccess){
		printf("Error solicitando memoria para d_R \n");
		return 1;
	}

	hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

	int blockSize = 32;
	dim3 dimGrid(ceil((colsB) / float(blockSize)), ceil((rowsA)/ float(blockSize)), 1);
	dim3 dimBlock(blockSize, blockSize, 1);

	MatrixMultiplyKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_R, colsA, rowsA, colsB, rowsB);
	hipMemcpy(h_R, d_R, sizeR, hipMemcpyDeviceToHost);

	print(h_A, rowsA, colsA);
	print(h_B, rowsB, colsB);
	print(h_R, rowsA, colsB);


	free(h_A);
	free(h_B);
	free(h_R);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_R);


	/* code */
	return 0;
}