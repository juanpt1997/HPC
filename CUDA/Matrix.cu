#include<stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include<time.h>
#include<hip/hip_runtime.h>

__global__
void matrizKernell(float* A, float* B, float* C, int rowsA, int colsA, int rowsB, int colsB){
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if((Row < rowsA) && (Col < colsB)){
        float sum = 0;
        for(int i=0; i < colsB; i++){
            sum += A[Row*colsA+i] * B[i*colsB+Col];
        }
        C[Row*colsB+Col] = sum;
    }
}

__host__
void print(float* M, int rows, int cols){
    printf("-----------MATRIX ------------- \n");
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            printf("%f ", M[i * cols + j]);
        }
        printf("\n");
    }
}

__host__
void receive(float* M, FILE* stream, int rows, int cols){
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            fscanf(stream, "%f", &M[i * cols +j]);
        }
    }
    fclose(stream);
}

int main(int argc, char** argv){
    if (argc != 3) {
        printf("Must be called with the names of the files \n");
        return 1;
    }

    float *A_in, *B_in, *C_out;
    int rowsA, colsA, rowsB, colsB;

    FILE *f1, *f2;
    f1 = fopen(argv[1], "r");
    f2 = fopen(argv[2], "r");

    fscanf(f1, "%d %d", &rowsA, &colsA);
    fscanf(f2, "%d %d", &rowsB, &colsB);
    
    //CPU
    A_in = (float*)malloc(rowsA * colsA * sizeof(float));
    B_in = (float*)malloc(rowsB * colsB * sizeof(float));
    C_out = (float*)malloc(rowsA * colsB * sizeof(float));
    
    receive(A_in, f1, rowsA, colsA);    
    receive(B_in, f2, rowsB, colsB);
    //print(A_in, rowsA, colsA);
    
    if(colsA != rowsB){
        printf("Debe ser igual el numero de las columnas de A, a las filas de B");
        return 1;
    }

    //GPU
    hipError_t error = hipSuccess;
    float *d_Ain, *d_Bin, *d_Cout;
    int blockSize = 32;
    //int gridSize = ceil((colsA*rowsA) / float(blockSize));
    dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid(ceil(colsA / float(blockSize)), ceil(rowsA / float(blockSize)), 1);

    error = hipMalloc((void**)&d_Ain, rowsA * colsA * sizeof(float));
    if(error != hipSuccess){
        printf("Error allocating memory d_Ain");
        return 1;
    }

    error = hipMalloc((void**)&d_Bin, rowsB * colsB * sizeof(float));
    if(error != hipSuccess){
        printf("Error allocating memory d_Bin");
        return 1;
    }

    error = hipMalloc((void**)&d_Cout, rowsA * colsB * sizeof(float));
    if(error != hipSuccess){
        printf("Error allocating memory d_Cout");
        return 1;
    }

    hipMemcpy(d_Ain, A_in, rowsA * colsA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Bin, B_in, rowsB * colsB * sizeof(float), hipMemcpyHostToDevice);

    matrizKernell<<<dimGrid, dimBlock>>>(d_Ain, d_Bin, d_Cout, rowsA, colsA, rowsB, colsB);
    //cudaDeviceSynchronize();

    hipMemcpy(C_out, d_Cout, rowsA * colsB * sizeof(float), hipMemcpyDeviceToHost);
    print(C_out, rowsA, colsB);
    
    free(A_in); free(B_in); free(C_out);
    hipFree(d_Ain); hipFree(d_Bin); hipFree(d_Cout);
    return 0;
}