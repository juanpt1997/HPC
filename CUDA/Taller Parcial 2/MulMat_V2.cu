#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void MulMatriz(float *m1, float *m2, float *mr, int fil1, int col1,int fil2, int col2) {
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int valor = 0;

	if(i<col2 && j<fil1){
		for(int k=0; k<fil2; k++){
			valor += m1[j*col1+k] * m2[k*col2+i];
		}
		mr[j*col2+i] = valor;
	}
}


__host__
void LeerMatriz(float* m1, float* m2, FILE* file, int fil1, int fil2, int col1, int col2) {
	for(int i=0; i<fil1*col1; i++){
		fscanf(file, "%f", &m1[i]);
    }

	for(int i=0; i<fil2*col2; i++){
		fscanf(file, "%f", &m2[i]);
    }

	fclose(file);
}

__host__
void EscribirMatriz(int fil, int col, float *m) { 
	FILE *f = fopen("outputIngenuo.txt", "a"); 
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			if(j==col-1){
				fprintf(f,"%f\n", m[i*col+j]); 
			}
			else{
				fprintf(f,"%f,", m[i*col+j]);
			}
		}
	}
	fprintf(f, "\n");
  	fclose(f); 
} 


int main(int argc, char** argv) {
	if (argc != 2) {
    	printf("Parametros incorrectos! \n");
    	return 1;
	}

	clock_t t_ini, t_fin; //Inicia reloj ------------------------
	double secs;
	t_ini = clock();

	int fil1, col1, fil2, col2;
	float *h_m1, *h_m2, *h_mr;
	float *d_m1, *d_m2, *d_mr;

	FILE *archivo;
	archivo = fopen(argv[1], "r");
	fscanf(archivo, "%d %d", &fil1, &col1);
	fscanf(archivo, "%d %d", &fil2, &col2);

	if (col1 != fil2){
		printf("No se pueden multiplicar matrices de estas dimensiones!");
		return 1;
	}

	int size1 = fil1*col1*sizeof(float); //tamaño en bits de cada matriz
	int size2 = fil2*col2*sizeof(float);
	int sizer = fil1*col2*sizeof(float);

	h_m1 = (float*)malloc(size1);
	h_m2 = (float*)malloc(size2);
	h_mr = (float*)malloc(sizer);
	hipMalloc(&d_m1, size1);
	hipMalloc(&d_m2, size2);
	hipMalloc(&d_mr, sizer);

	int blockSize = 32;
	dim3 dimBlock(blockSize, blockSize, 1);
	dim3 dimGrid(ceil(col1/float(blockSize)), ceil(col1/float(blockSize)), 1);

	LeerMatriz(h_m1, h_m2, archivo, fil1, fil2, col1, col2); 

	hipMemcpy(d_m1, h_m1, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_m2, h_m2, size2, hipMemcpyHostToDevice);

	MulMatriz<<<dimGrid, dimBlock>>>(d_m1, d_m2, d_mr, fil1, col1, fil2, col2); //Ejecución del kernel
	hipMemcpy(h_mr, d_mr, sizer, hipMemcpyDeviceToHost); 

	EscribirMatriz(fil1, col1, h_m1);
	EscribirMatriz(fil2, col2, h_m2);
	EscribirMatriz(fil1, col2, h_mr);

	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_mr);
	free(h_m1);
	free(h_m2);
	free(h_mr);

  	t_fin = clock(); //Fin reloj ------------------------
  	secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  	printf("Tiempo de ejecucion: %.16g milisegundos\n", secs * 1000.0);
  	
  	return 0;
}
