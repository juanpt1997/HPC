#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define TILE_DIM 32

__global__ void MulMatriz(float *m1, float *m2, float *mr, int fil1, int col1,int fil2, int col2) {
	
	int i = blockIdx.y*TILE_DIM + threadIdx.y; // Row
	int j = blockIdx.x*TILE_DIM + threadIdx.x; // Col

	__shared__ float m1s[TILE_DIM][TILE_DIM];
	__shared__ float m2s[TILE_DIM][TILE_DIM];

	int valor = 0;
	
    int n = 0, m = 0;
    while(m < gridDim.x && n < gridDim.y){

		if(((m*TILE_DIM) + threadIdx.x)<col1 && i<fil1) {
			m1s[threadIdx.y][threadIdx.x] = m1[(i*col1) + ((m*TILE_DIM) + threadIdx.x)];
		}
		else {
			m1s[threadIdx.y][threadIdx.x] = 0;
		}
		
		if((n*TILE_DIM + threadIdx.y)<fil2 && j<col2) {
			m2s[threadIdx.y][threadIdx.x] = m2[((n*TILE_DIM + threadIdx.y)*col2) + j];
		}
		else {
			m2s[threadIdx.y][threadIdx.x] = 0;
		}
		m++; 
		n++;

		__syncthreads();

		for (int k=0; k < TILE_DIM ; k++) {
			valor += m1s[threadIdx.y][k]*m2s[k][threadIdx.x];
		}
		__syncthreads();
	}

	if(i<fil1 && j<col2) {
		mr[(i*col2) + j] = valor;
	}
}


__host__
void LeerMatriz(float* m1, float* m2, FILE* file, int fil1, int fil2, int col1, int col2) {
	for(int i=0; i<fil1*col1; i++){
		fscanf(file, "%f", &m1[i]);
    }

	for(int i=0; i<fil2*col2; i++){
		fscanf(file, "%f", &m2[i]);
    }

	fclose(file);
}

__host__
void EscribirMatriz(int fil, int col, float *m) { 
	FILE *f = fopen("outputSH.txt", "a"); 
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			if(j==col-1){
				fprintf(f,"%f\n", m[i*col+j]); 
			}
			else{
				fprintf(f,"%f,", m[i*col+j]);
			}
		}
	}
	fprintf(f, "\n");
  	fclose(f); 
} 


int main(int argc, char** argv) {
	if (argc != 2) {
    	printf("Parametros incorrectos! \n");
    	return 1;
	}

	clock_t t_ini, t_fin; //Inicia reloj ------------------------
	double secs;
	t_ini = clock();

	int fil1, col1, fil2, col2;
	float *h_m1, *h_m2, *h_mr;
	float *d_m1, *d_m2, *d_mr;

	FILE *archivo;
	archivo = fopen(argv[1], "r");
	fscanf(archivo, "%d %d", &fil1, &col1);
	fscanf(archivo, "%d %d", &fil2, &col2);

	if (col1 != fil2){
		printf("No se pueden multiplicar matrices de estas dimensiones!");
		return 1;
	}

	int size1 = fil1*col1*sizeof(float); //tamaño en bits de cada matriz
	int size2 = fil2*col2*sizeof(float);
	int sizer = fil1*col2*sizeof(float);

	h_m1 = (float*)malloc(size1);
	h_m2 = (float*)malloc(size2);
	h_mr = (float*)malloc(sizer);
	hipMalloc(&d_m1, size1);
	hipMalloc(&d_m2, size2);
	hipMalloc(&d_mr, sizer);

	int blockSize = 32;
	dim3 dimBlock(blockSize, blockSize, 1);
	dim3 dimGrid(ceil(col1/float(blockSize)), ceil(col1/float(blockSize)), 1);

	LeerMatriz(h_m1, h_m2, archivo, fil1, fil2, col1, col2); 

	hipMemcpy(d_m1, h_m1, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_m2, h_m2, size2, hipMemcpyHostToDevice);

	MulMatriz<<<dimGrid, dimBlock>>>(d_m1, d_m2, d_mr, fil1, col1, fil2, col2); //Ejecución del kernel
	hipMemcpy(h_mr, d_mr, sizer, hipMemcpyDeviceToHost); 

	EscribirMatriz(fil1, col1, h_m1);
	EscribirMatriz(fil2, col2, h_m2);
	EscribirMatriz(fil1, col2, h_mr);

	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_mr);
	free(h_m1);
	free(h_m2);
	free(h_mr);

  	t_fin = clock(); //Fin reloj ------------------------
  	secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  	printf("Tiempo de ejecucion: %.16g milisegundos\n", secs * 1000.0);
  	
  	return 0;
}
