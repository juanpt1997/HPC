#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void MulMatriz(float *min, float *mout, int fil, int col)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ((i < fil) && (j < col)){
        mout[i*col+j] = 2*min[i*col+j]; 
    }
}


int main()
{
	//Inicia reloj ------------------------
	clock_t t_ini, t_fin;
  	double secs;
  	t_ini = clock();
  	//-------------------------------------

	int fil, col;
	float *h_min, *h_mout;
	float *d_min, *d_mout;

	fil = 5;
	col = 6; //con el más grande se hace la referencia para la matriz en 1D

	int size = fil*col*sizeof(float); //tamaño en bits de cada matriz

	h_min = (float*)malloc(size);
	h_mout = (float*)malloc(size);
	hipMalloc(&d_min, size);
    hipMalloc(&d_mout, size);

    int blockSize = 32;
    dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid(ceil(col/float(blockSize)), ceil(col/float(blockSize)), 1);

	//Iniciar matriz con valor 13------------------
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			h_min[i*col+j] = 13; 
		}
	}

	//Imprimir resultados------------------
	printf("matriz: ----------------------\n"); 
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			printf("%f ", h_min[i*col+j]);
		}
		printf("\n"); 
	}	

	printf("\nmatriz x5: ----------------------\n"); 

	hipMemcpy(d_min, h_min, size, hipMemcpyHostToDevice);
	MulMatriz<<<dimGrid, dimBlock>>>(d_min, d_mout, fil, col); //Ejecución del kernel
	hipMemcpy(h_mout, d_mout, size, hipMemcpyDeviceToHost); //Copia de datos al host
	
	//Imprimir resultados------------------
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			printf("%f ", h_mout[i*col+j]);
		}
		printf("\n"); 
	}
	//-------------------------------------

	hipFree(d_min);
    hipFree(d_mout);
	free(h_min);
	free(h_mout);

	//Fin reloj ------------------------
  	t_fin = clock();
  	secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  	printf("Tiempo de ejecucion: %.16g milisegundos\n", secs * 1000.0);
  	
  	return 0;
}
